#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <hip/hiprtc.h>
#include <jitify.hpp>

// #include "types.h.jit"
#include "types.hpp.jit"
#include "timestamps.hpp.jit"

const char* kernel =
R"***(
#define _LIBCUDACXX_USE_CXX20_CHRONO
#define _LIBCUDACXX_USE_CXX17_TYPE_TRAITS
#include <simt/chrono>
#include <cudf/types.hpp>
#include <cudf/wrappers/timestamps.hpp>
template<int N, typename T>
__global__ void kernel(T* data) {}
)***";

int main(void) {

  const std::vector<std::string> headers{cudf_types_hpp, cudf_wrappers_timestamps_hpp};

  static jitify::JitCache kernel_cache;
  jitify::Program program = kernel_cache.program(kernel, headers, {
    "-std=c++14",
    // define libcudacxx jitify guards
    "-D_LIBCUDACXX_HAS_NO_CTIME",
    "-D_LIBCUDACXX_HAS_NO_WCHAR",
    "-D_LIBCUDACXX_HAS_NO_CFLOAT",
    "-D_LIBCUDACXX_HAS_NO_STDINT",
    "-D_LIBCUDACXX_HAS_NO_CSTDDEF",
    "-D_LIBCUDACXX_HAS_NO_CLIMITS",
    "-D_LIBCPP_DISABLE_VISIBILITY_ANNOTATIONS",
    // "-I/usr/include/linux",
    "-I/home/ptaylor/dev/rapids/jitify-libcu++-test/thirdparty/libcudacxx/include",
    "-I/home/ptaylor/dev/rapids/jitify-libcu++-test/thirdparty/libcudacxx/libcxx/include"
  });

}
