#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <hip/hiprtc.h>
#include <jitify.hpp>

#include "types.hpp.jit"
#include "timestamps.hpp.jit"
#include "operation.h.jit"
#include "traits.h.jit"

const char* kernel =
R"***(
// #include <cstdint>
// #include <type_traits>

#include <cudf/types.hpp>
#include <simt/limits>
#include <cudf/wrappers/timestamps.hpp>

// problematic
// #include "operation.h"
template<int N, typename T>
__global__ void kernel(T* data) {}
)***";

int main(void) {

  const std::vector<std::string> headers{
    cudf_types_hpp,
    cudf_wrappers_timestamps_hpp,
    operation_h,
    traits_h
  };

  static jitify::JitCache kernel_cache;
  jitify::Program program = kernel_cache.program(kernel, headers, {
    "-std=c++14",
    "-D__CUDACC_RTC__",
    "-D__CHAR_BIT__=" + std::to_string(__CHAR_BIT__),
    // define libcudacxx jitify guards
    "-D_LIBCUDACXX_HAS_NO_CTIME",
    "-D_LIBCUDACXX_HAS_NO_WCHAR",
    "-D_LIBCUDACXX_HAS_NO_CFLOAT",
    "-D_LIBCUDACXX_HAS_NO_STDINT",
    "-D_LIBCUDACXX_HAS_NO_CSTDDEF",
    "-D_LIBCUDACXX_HAS_NO_CLIMITS",
    "-D_LIBCPP_DISABLE_VISIBILITY_ANNOTATIONS",
    "-I/home/ptaylor/dev/rapids/jitify-libcu++-test/thirdparty/libcudacxx/include",
  });

}
