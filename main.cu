#include "hip/hip_runtime.h"
#include <string>
#include <vector>
#include <hip/hiprtc.h>
#include <jitify.hpp>

// #include "timestamps.hpp"
#include "timestamps.hpp.jit"

const char* kernel =
R"***(
  #include <cudf/wrappers/timestamps.hpp>
  template<int N, typename T>
  __global__ void kernel(T* data) {
  }
)***";

int main(void) {

  const std::vector<std::string> headers{cudf_wrappers_timestamps_hpp};

  static jitify::JitCache kernel_cache;
  jitify::Program program = kernel_cache.program(kernel, headers, {
    "-std=c++14",
    "-D__x86_64__",
    "-D_LIBCPP_STD_VER=14",
    "-D_LIBCPP_HAS_NO_PRAGMA_PUSH_POP_MACRO",
    "-D_LIBCPP_DISABLE_VISIBILITY_ANNOTATIONS",
    "-I/usr/include",
    "-I/usr/include/c++/7",
    "-I/usr/include/c++/7/tr1",
    "-I/usr/include/x86_64-linux-gnu",
    "-I/usr/local/cuda-10.1/targets/x86_64-linux/include",
    "-I/home/ptaylor/dev/rapids/jitify-libcu++-test/thirdparty/libcudacxx/include",
    "-I/home/ptaylor/dev/rapids/jitify-libcu++-test/thirdparty/libcudacxx/libcxx/include"
  });

}
